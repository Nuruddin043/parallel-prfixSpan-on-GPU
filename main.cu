#include "hip/hip_runtime.h"

#include<string>
#include <sstream>
#include<ctime>
#include <chrono>
#include<fstream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>
#include<stack>
#include<vector>
#include "frequent_items.h"
#include "projected_database.h"
#include "prefix_span.h"
using namespace std;
fstream _file;


vector < vector<int> > sequential_patterns;

int main() {
	int found, cnt = 0, total_row = 0;
	string line, temp;
	clock_t startt,endt;
	thrust::host_vector<int, thrust::cuda::experimental::pinned_allocator<int>> Hdata;
	thrust::host_vector<int, thrust::cuda::experimental::pinned_allocator<int>> Hstart;
	thrust::host_vector<int, thrust::cuda::experimental::pinned_allocator<int>> Hend;
	thrust::device_vector<int> device_data;
	thrust::device_vector<int> start(total_row);
	thrust::device_vector<int> end(total_row);
	freopen("data.out","w",stdout);

	ifstream file;
	file.open("MSNBC_SPMF.txt");
	if (!file) {
		cout << "file not found \n";
		return -1;
	}
//	auto startt = chrono::steady_clock::now();
	while (getline(file, line))
	{
		istringstream ss(line);
	//	int* ptr = raw_pointer_cast(&data[total_row]);
		Hstart.push_back(cnt);
		while (ss >> temp)
		{
			if (stringstream(temp) >> found) {
				Hdata.push_back(found);
				cnt++;
			}

		}
		total_row++;

		Hend.push_back(cnt);

	}
	file.close();


	device_data=Hdata;
	start=Hstart;
	end=Hend;
	int* dptr = raw_pointer_cast(&device_data[0]);
	int* startPtr = raw_pointer_cast(&start[0]);
	int* endPtr = raw_pointer_cast(&end[0]);
	startt = clock();
	prefix_Span(dptr, startPtr, endPtr, total_row,sequential_patterns);

//	auto endt = chrono::steady_clock::now();
//	auto diff = endt - startt;
	//cout << chrono::duration <double, milli>(diff).count() << " ms" << endl;
	endt = clock();
  double interval = (double)(endt - startt) / CLOCKS_PER_SEC;
  cout<<"cost time : " <<interval<<endl;
	for (int i = 0; i < sequential_patterns.size(); i++) {
		for (int j = 0; j < sequential_patterns[i].size(); j++) {
			cout << sequential_patterns[i][j] << " ";
		}
		cout << endl;
	}
	 	gpuErrchk(hipDeviceReset());
}
